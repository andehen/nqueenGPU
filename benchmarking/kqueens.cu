#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define NUM_BLOCKS $NUM_BLOCKS
#define NUM_THREADS $NUM_THREADS
#define K $K
#define MAX_ITER $MAX_ITER 

using namespace std;

__device__ int checkDiagonals(int q,int i, int* S)
{
	int I = blockIdx.x*NUM_THREADS*K + threadIdx.x*K;
	int j = 1;
	for (j; j<=i; j++){
		if (S[I+i-j] == q-j | S[I+i-j] == q+j){
			return 0;
		} 		
	}	
	return 1;
}

__device__ int sum(int row[], int len)
{
	int i = 0;
	int s = 0;
	for (i; i<len; i++){
		s += row[i];
	}
	return s;
}

__global__ void setup_kernel (hiprandState * state, unsigned long seed)
{
	int id = blockIdx.x*NUM_BLOCKS + threadIdx.x;
	hiprand_init ( seed, id, 0, &state[id] );
}

__global__ void kernel(int* S, hiprandState* globalState)
{
	int I = blockIdx.x*NUM_THREADS*K + threadIdx.x*K;
	int ind = blockIdx.x*NUM_BLOCKS + threadIdx.x;
	int D[K];				
	int N[K][K];			
	
	int i = 0;				
	int j = 0;

	int q; 
	
	for (i; i<K; i++){
		S[I+i] = -1;
		D[i] = 0;
		for (j;j<K;j++){
			N[i][j] = 0;
		}
		j = 0;
	}

	i = 0;

	int iter = 0;

	hiprandState localState = globalState[ind];
	
	while (iter < MAX_ITER){
	
		q = hiprand_uniform( &localState ) * K;
		
		if (D[q] == 0 & N[i][q] == 0){ 		 
			N[i][q] = 1;
			if (checkDiagonals(q,i,S)==1){	
				S[I+i] = q;			
				D[q] = 1;
				i++;
				if (i==K){			
					break;
				}
			}
		}
		if (sum(N[i],K) + sum(D,K) == K){
			D[S[I+i-1]] = 0;
			S[I+i-1] = -1;
			j = 0;
			for (j;j<K;j++){	
				N[i][j] = 0;
			}		
			i--;				
		}
		iter++;
	}
}

int main() 
{
	hiprandState* devStates;
	hipMalloc ( &devStates, K*sizeof( hiprandState ) );

	setup_kernel <<< NUM_BLOCKS, NUM_THREADS>>> ( devStates,unsigned(time(NULL)) );

	int solution_host[K*NUM_BLOCKS*NUM_THREADS];
	int* solution_dev;

	hipMalloc((void**) &solution_dev, sizeof(int)*K*NUM_BLOCKS*NUM_THREADS);

	clock_t begin = clock();
	kernel<<<NUM_BLOCKS,NUM_THREADS>>> (solution_dev, devStates);
	hipMemcpy(solution_host, solution_dev, sizeof(int)*K*NUM_BLOCKS*NUM_THREADS, hipMemcpyDeviceToHost);
	clock_t end = clock();
	
	double elapsed_sec = double(end - begin)/(CLOCKS_PER_SEC/1000);

	cout << elapsed_sec << endl;

	int solution_count = 0;
	for (int l=0;l<NUM_BLOCKS*NUM_THREADS;l++){
		if (solution_host[l*K+K-1]!=-1){
			solution_count++;
		}
	}
	cout << solution_count << endl;

	hipFree(devStates);
	hipFree(solution_dev);
	
	hipDeviceReset(); 

	return 0;


}
