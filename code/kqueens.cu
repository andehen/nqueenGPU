#include <stdio.h>
#include <iostream>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define k 110 // set problem size
#define NUM_BLOCKS 32
#define NUM_THREADS 512
#define MAX_ITER 2000

using namespace std;

__device__ float generate(hiprandState* globalState, int ind2)
// Function to generate random number in thread
{
	int ind = (blockIdx.x+1)*threadIdx.x;
	hiprandState localState = globalState[ind];
	float rnd = hiprand_uniform( &localState );
	globalState[ind] = localState;
	return rnd;
}

__device__ int checkDiagonals(int q,int i, int* S)
// Returns 1 if no queen in diagonal, else 0
{
	int I = blockIdx.x*NUM_THREADS*k + threadIdx.x*k;
	int j = 1;
	for (j; j<=i; j++){
		if (S[I+i-j] == q-j | S[I+i-j] == q+j){
			return 0;
		} 		
	}	
	return 1;
}

__device__ int checkSolution(int S[]){
	for (int i=0;i<k;i++){
		if (S[i]==-1){
			return 0;
		}
	}
	return 1;
}

__device__ int sum(int row[], int len)
// Returns sum of an array 
{
	int i = 0;
	int s = 0;
	for (i; i<len; i++){
		s += row[i];
	}
	return s;
}

__global__ void setup_kernel (hiprandState * state, unsigned long seed)
{
	int id = (blockIdx.x+1)*threadIdx.x;
	hiprand_init ( seed, id, 0, &state[id] );
}

__global__ void kernel(int* S, hiprandState* globalState)
{
	//__shared__ int S_shared[NUM_BLOCKS*NUM_THREADS*k];
	int I = blockIdx.x*NUM_THREADS*k + threadIdx.x*k;
	// Initialize varaibles
	//int S[k]; 				// Holds current solution
	int D[k];				// Rows where queens is placed
	int N[k][k];				// Positions tried at column i
	
	int i = 0;				
	int j = 0;

	int q; 
	
	// Set to start values
	for (i; i<k; i++){
		S[I+i] = -1;
		D[i] = 0;
		for (j;j<k;j++){
			N[i][j] = 0;
		}
		j = 0;
	}

	i = 0;

	int iter = 0;
	
	while (iter < MAX_ITER){
	
		q = (generate(globalState, i) * k);	// Generate random number
		
		if (D[q] == 0 & N[i][q] == 0){ 		// Row clear and not tried before 
			N[i][q] = 1;
			if (checkDiagonals(q,i,S)==1){	// If no attacking queens in diagonal
				S[I+i] = q;			// it can proceed
				D[q] = 1;
				i++;
				if (i==k){			// Finished!
					break;
				}
			}
		}
		if (sum(N[i],k) + sum(D,k) == k){
			D[S[I+i-1]] = 0;
			S[I+i-1] = -1;
			j = 0;
			for (j;j<k;j++){		// Reset N
				N[i][j] = 0;
			}		
			i--;				// Backtrack
		}
		iter++;
	}
	// For now, just print solution for each thread for debugging
//	if (checkSolution(S)==1){
//		printf("Sol from block %d, thread %d: ", blockIdx, threadIdx);
//		for (int l=0;l<k;l++){
//			printf("%d ", S[l]);
//		}
//		printf("\n");
//	}

	//for (int p=0;p<k;p++){
	//	solution[I+p] = S[p];
	//}
}

int main() 
{
//	size_t avail;
//	size_t total;
//	cudaMemGetInfo( &avail, &total );
//	size_t used = total - avail;
//	cout << "Device memory used: " << used << endl;

	hiprandState* devStates;
	hipMalloc ( &devStates, k*sizeof( hiprandState ) );

	// Initialze seeds
	setup_kernel <<< NUM_BLOCKS, NUM_THREADS>>> ( devStates,unsigned(time(NULL)) );

	int solution_host[k*NUM_BLOCKS*NUM_THREADS];
	int* solution_dev;

	hipMalloc((void**) &solution_dev, sizeof(int)*k*NUM_BLOCKS*NUM_THREADS);

	clock_t begin = clock();
	kernel<<<NUM_BLOCKS,NUM_THREADS>>> (solution_dev, devStates);
	hipMemcpy(solution_host, solution_dev, sizeof(int)*k*NUM_BLOCKS*NUM_THREADS, hipMemcpyDeviceToHost);
	clock_t end = clock();
	
	double elapsed_sec = double(end - begin)/CLOCKS_PER_SEC;

	cout << "Time: " << elapsed_sec << endl;

	int solution_count = 0;
	for (int l=0;l<NUM_BLOCKS*NUM_THREADS;l++){
		if (solution_host[l*k+k-1]!=-1){
//			for (int p=0;p<k;p++){
//				printf("%d ", solution_host[l*k+p]);
//			}
			//printf("\n");
			solution_count++;
		}
	}
	printf("%d\n", solution_count);

	// Free memory
	hipFree(devStates);
	hipFree(solution_dev);
	
	hipDeviceReset(); // Tried to fix memory leakage

	return 0;


}
